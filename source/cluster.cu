#include "hip/hip_runtime.h"
#include "cluster.h"

#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <stdio.h>
#include <stdint.h>
#include <float.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <limits>

//Represents a cluster
struct Cluster
{
	float lower;
	float upper;
	uint32_t index;
};

//Comparison operator for sorting the list of duplicate merges in descending order of LHS cluster index
class MergesDescendingIndexComp
{
	public:
		__device__ bool operator()(const Merge& lhsMerge, const Merge& rhsMerge) {
			return lhsMerge.lhs > rhsMerge.lhs;
		}
};

//Predicate to identify merged clusters that can be removed from the array
class IsMergedPredicate
{
	public:
		__device__ bool operator()(const Cluster& c) {
			return c.lower == HUGE_VALF;
		}
};

//Builds the array of initial clusters, where each data point is its own cluster
__global__ void buildInitialClusters(float* values, const uint32_t N, Cluster* clusters)
{
	uint32_t thisThreadId = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (thisThreadId < N)
	{
		Cluster c;
		c.lower = c.upper = values[thisThreadId];
		c.index = thisThreadId;
		clusters[thisThreadId] = c;
	}
}

//Computes the distances between a set of clusters, using the specified linkage metric
__global__ void computeDistances(Cluster* clusters, const uint32_t N, float* distances, LinkageType linkage)
{
	uint32_t thisThreadId = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (thisThreadId < (N - 1))
	{
		if (linkage == CompleteLinkage)
		{
			//Use complete-linkage
			distances[thisThreadId] = clusters[thisThreadId + 1].upper - clusters[thisThreadId].lower;
		}
		else
		{
			//Use single-linkage
			distances[thisThreadId] = clusters[thisThreadId + 1].lower - clusters[thisThreadId].upper;
		}
	}
}

//Merges all clusters with a distance of zero, ignoring collisions
__global__ void mergeDuplicates(Cluster* clusters, const uint32_t N, float* distances, Merge* merges, uint32_t* mergeIndex)
{
	uint32_t thisThreadId = threadIdx.x + blockDim.x * blockIdx.x;

	if (thisThreadId < (N - 1) && distances[thisThreadId] == 0)
	{
		//Add the dendrogram node to the list of merges
		uint32_t index = atomicAdd(mergeIndex, 1);
		merges[index].lhs = clusters[thisThreadId].index;
		merges[index].rhs = clusters[thisThreadId + 1].index;
		merges[index].distance = 0;

		//Flag the right-hand cluster for removal from the array
		clusters[thisThreadId + 1].lower = HUGE_VALF;
		clusters[thisThreadId + 1].upper = HUGE_VALF;
	}
}

//Given the current minimum distance, flags all cluster pairs with that distance to be merged
__global__ void markClustersForMerge(float* distances, const uint32_t N, const float* min, bool* shouldMerge)
{
	uint32_t thisThreadId = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (thisThreadId < (N - 1)) {
		shouldMerge[thisThreadId] = (distances[thisThreadId] == *min);
	}	
}

//Flag all detected collisions
__global__ void flagDetectedCollisions(const uint32_t N, bool* shouldMerge, bool* collisionFlags)
{
	uint32_t thisThreadId = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (thisThreadId < (N - 1))
	{
		//Reset the collision flag for each thread
		collisionFlags[thisThreadId] = false;
		
		//Set the new collision flag state if a collision is detected
		if (thisThreadId > 0 && shouldMerge[thisThreadId] == true && shouldMerge[thisThreadId - 1] == true) {
			collisionFlags[thisThreadId] = true;
		}
	}	
}

//Identify the indices of the first collision in each contiguous set of collisions
__global__ void identifyCollisionLeaders(const uint32_t N, bool* shouldMerge, bool* collisionFlags, uint32_t* collisionLeaders)
{
	uint32_t thisThreadId = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (thisThreadId < (N - 1))
	{
		collisionLeaders[thisThreadId] = 0;
		if (thisThreadId > 0 && collisionFlags[thisThreadId] == true && collisionFlags[thisThreadId - 1] == false) {
			collisionLeaders[thisThreadId] = thisThreadId;
		}
	}
}

//Performs the set of merges specified by the given set of merge flags
__global__ void performMerges(Cluster* clusters, const uint32_t N, float* distances, bool* shouldMerge, Merge* merges, uint32_t* mergeIndex, bool* collisionFlags, uint32_t* collisionLeaders)
{
	uint32_t thisThreadId = threadIdx.x + blockDim.x * blockIdx.x;
	
	//Fix collisions to ensure a left-to-right ordering of merges
	if (thisThreadId < (N - 1) && collisionFlags[thisThreadId] == true)
	{
		uint32_t collisionsStart = collisionLeaders[thisThreadId];
		shouldMerge[thisThreadId] = ((thisThreadId - collisionsStart) % 2 == 1);
	}
	
	if (thisThreadId < (N - 1) && shouldMerge[thisThreadId] == true)
	{
		//Add the dendrogram node to the list of merges
		uint32_t index = atomicAdd(mergeIndex, 1);
		merges[index].lhs = clusters[thisThreadId].index;
		merges[index].rhs = clusters[thisThreadId + 1].index;
		merges[index].distance = distances[thisThreadId];
		
		//Update the cluster bounds for the merged cluster
		clusters[thisThreadId].upper = clusters[thisThreadId + 1].upper;
		
		//Flag the right-hand cluster for removal from the array
		clusters[thisThreadId + 1].lower = HUGE_VALF;
		clusters[thisThreadId + 1].upper = HUGE_VALF;
	}
}

#define NUMBLOCKS(numValues,numThreadsPerBlock) ceil((float)numValues / (float)numThreadsPerBlock)

Merge* performClustering(float* values, uint32_t numValues, LinkageType linkage)
{
	//Determine the number of threads that can be run per block
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	auto numThreadsPerBlock = prop.maxThreadsPerBlock;
	
	//Copy the values to the GPU
	float* dValues;
	hipMalloc(&dValues, numValues * sizeof(float));
	hipMemcpy(dValues, values, numValues * sizeof(float), hipMemcpyHostToDevice);
	
	//Sort the values on the GPU
	thrust::sort(thrust::device, dValues, dValues + numValues);
	hipDeviceSynchronize();
	
	//Copy the sorted values back to the CPU so the caller has access to them
	hipMemcpy(values, dValues, numValues * sizeof(float), hipMemcpyDeviceToHost);
	
	//Allocate the array for the list of merges on the GPU
	Merge* dMerges;
	hipMalloc(&dMerges, (numValues - 1) * sizeof(Merge));
	
	//Allocate the boolean array of merge flags on the GPU
	bool* dShouldMerge;
	hipMalloc(&dShouldMerge, (numValues-1) * sizeof(bool));
	hipMemset(dShouldMerge, 0, (numValues-1) * sizeof(bool));
	
	//Allocate the boolean array of collision flags on the GPU
	bool* dCollisionFlags;
	hipMalloc(&dCollisionFlags, (numValues-1) * sizeof(bool));
	hipMemset(dCollisionFlags, 0, (numValues-1) * sizeof(bool));
	
	//Allocate the array for contiguous collision block leader indices on the GPU
	uint32_t* dCollisionLeaders;
	hipMalloc(&dCollisionLeaders, (numValues - 1) * sizeof(uint32_t));
	
	//Allocate the array of cluster distances on the GPU
	float* dDistances;
	hipMalloc(&dDistances, (numValues - 1) * sizeof(float));
	
	//Allocate the merge counter on the GPU
	uint32_t* dMergeIndex;
	hipMalloc(&dMergeIndex, sizeof(uint32_t));
	hipMemset(dMergeIndex, 0, sizeof(uint32_t));
	
	//Build the list of initial clusters on the GPU
	Cluster* dClusters;
	hipMalloc(&dClusters, numValues * sizeof(Cluster));
	buildInitialClusters<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(dValues, numValues, dClusters);
	hipDeviceSynchronize();
	
	//Compute the distances between the initial set of clusters
	computeDistances<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(dClusters, numValues, dDistances, linkage);
	hipDeviceSynchronize();
	
	//Merge duplicates in parallel
	mergeDuplicates<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(dClusters, numValues, dDistances, dMerges, dMergeIndex);
	hipDeviceSynchronize();
	
	//Copy the current merge counter back to the CPU
	uint32_t hMergeIndex;
	hipMemcpy(&hMergeIndex, dMergeIndex, sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	//Sort the merges in descending order of LHS index, so that the first value in each block of duplicates is the one clustered last
	thrust::sort(thrust::device, dMerges, dMerges + hMergeIndex, MergesDescendingIndexComp());
	hipDeviceSynchronize();
	
	//Remove the clusters that were flagged for removal
	uint32_t currentArrayLength = numValues;
	Cluster* newArrayEnd = thrust::remove_if(thrust::device, dClusters, dClusters + currentArrayLength, IsMergedPredicate());
	currentArrayLength = (newArrayEnd - dClusters);
	hipDeviceSynchronize();
	
	//Perform clustering
	do
	{
		//Compute the distances between the current set of clusters
		computeDistances<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(dClusters, currentArrayLength, dDistances, linkage);
		hipDeviceSynchronize();
		
		//Find the minimum distance value
		float* minDistance = thrust::min_element(thrust::device, dDistances, dDistances + (currentArrayLength-1));
		
		//Identify which clusters should be merged in this iteration
		markClustersForMerge<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(dDistances, currentArrayLength, minDistance, dShouldMerge);
		hipDeviceSynchronize();
		
		//Identify collisions and set the values of the collision flags array accordingly
		flagDetectedCollisions<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(currentArrayLength, dShouldMerge, dCollisionFlags);
		hipDeviceSynchronize();
		
		//Identify the leading index for each set of contiguous collisions
		thrust::fill(thrust::device, dCollisionLeaders, dCollisionLeaders + currentArrayLength, 0);
		identifyCollisionLeaders<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(currentArrayLength, dShouldMerge, dCollisionFlags, dCollisionLeaders);
		hipDeviceSynchronize();
		
		//Propagate leader indices to adjacent array slots
		thrust::inclusive_scan(thrust::device, dCollisionLeaders, dCollisionLeaders + currentArrayLength, dCollisionLeaders, thrust::maximum<uint32_t>());
		hipDeviceSynchronize();
		
		//Perform the merges for this iteration, using the complete-linkage version of the kernel
		performMerges<<< numThreadsPerBlock, NUMBLOCKS(numValues,numThreadsPerBlock) >>>(dClusters, currentArrayLength, dDistances, dShouldMerge, dMerges, dMergeIndex, dCollisionFlags, dCollisionLeaders);
		hipDeviceSynchronize();
		
		//Remove the clusters that were flagged for removal
		Cluster* newArrayEnd = thrust::remove_if(thrust::device, dClusters, dClusters + currentArrayLength, IsMergedPredicate());
		currentArrayLength = (newArrayEnd - dClusters);
		hipDeviceSynchronize();
		
	} while (currentArrayLength > 1);
	
	//Copy the list of merges from the GPU to the CPU
	Merge* hMerges = (Merge*)(malloc((numValues - 1) * sizeof(Merge)));
	hipMemcpy(hMerges, dMerges, (numValues - 1) * sizeof(Merge), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	//Free allocated GPU memory
	hipFree(dValues);
	hipFree(dMerges);
	hipFree(dClusters);
	hipFree(dShouldMerge);
	hipFree(dCollisionFlags);
	hipFree(dCollisionLeaders);
	hipFree(dDistances);
	hipFree(dMergeIndex);
	
	//Return the list of merges (caller is responsible for freeing the memory)
	return hMerges;
}

void freeDendrogram(Merge* merges) {
	free(merges);
}

GPUDetails getGpuDetails()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	
	GPUDetails details;
	details.name = string(prop.name);
	details.maxThreads = prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount;
	details.clockSpeedGhz = (double)(prop.clockRate) / 1000.0 / 1000.0;
	return details;
}
